#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_SIZE 128
#define BANK_SIZE 32

__global__ void padArray(int* array) {
// Shared memory with padding
    __shared__ int sharedArray[ARRAY_SIZE + ARRAY_SIZE / BANK_SIZE];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int index = bid * blockDim.x + tid;

// Load data into shared memory with padding
    sharedArray[tid] = array[index];
    __syncthreads();

// Access all keys from the original bank 0 in one clock pulse
    int offset = tid / BANK_SIZE;
    int newIndex = tid + offset;

// Use the modified index for accessing the padded shared memory
    int result = sharedArray[newIndex];

// Print the result for demonstration
    printf("Thread %d: Original Value: %d, Padded Value: %d\n", tid, array[index], result);
}

int main() {
    int array[ARRAY_SIZE];

// Initialize array values (you can replace this with your data)
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        array[i] = i * 2;
    }

    int* d_array;

// Allocate device memory
    hipMalloc((void**)&d_array, ARRAY_SIZE * sizeof(int));

// Copy array from host to device
    hipMemcpy(d_array, array, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

// Define block and grid dimensions
    dim3 blockDim(BANK_SIZE);
    dim3 gridDim((ARRAY_SIZE + blockDim.x - 1) / blockDim.x);

// Launch kernel
    padArray<<<gridDim, blockDim>>>(d_array);

// Synchronize device to ensure print statements are displayed
    hipDeviceSynchronize();

// Free allocated memory
    hipFree(d_array);

    return 0;
}